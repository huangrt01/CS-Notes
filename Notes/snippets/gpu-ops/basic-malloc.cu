#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void initWith(float num, float *a, int N)
{

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;
  float *h_c;

  hipMalloc(&a, size);
  hipMalloc(&b, size);
  hipMalloc(&c, size);
  hipHostMalloc(&h_c, size);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  /*
   * Create 3 streams to run initialize the 3 data vectors in parallel.
   */

  hipStream_t stream1, stream2, stream3;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);

  /*
   * Give each `initWith` launch its own non-standard stream.
   */

  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(0, c, N);

  const int numberOfSegments = 4;
  int segmentN = N / numberOfSegments;             
  size_t segmentSize = size / numberOfSegments;

  for(int i = 0; i < numberOfSegments; i++){
    int segmentOffset = i * segmentN;
    hipStream_t stream;
    hipStreamCreate(&stream);
    addVectorsInto<<<numberOfBlocks, threadsPerBlock, 0, stream>>>(c[segmentOffset], a[segmentOffset], b[segmentOffset], segmentSize);
    hipMemcpyAsync(&h_c[segmentOffset], &c[segmentOffset], segmentSize, hipMemcpyDeviceToHost, stream);
    hipStreamDestroy(stream);
  }

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  checkElementsAre(7, h_c, N);

  /*
   * Destroy streams when they are no longer needed.
   */

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);

  hipFree(a);
  hipFree(b);
  hipFree(c);
  hipHostFree(h_c);
}
