#include "hip/hip_runtime.h"
#include <stdio.h>

#define N  64

__global__ void sharedMatMult( float * a, float * b, float *c){
  __shared__ float aTile[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float bTile[BLOCK_SIZE][BLOCK_SIZE];

  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  float sum = 0.0f;

  for(int k = 0; k < N; k += BLOCK_SIZE){
    aTile[threadIdx.x][threadIdx.y] = a[row * N + threadIdx.y + k];
    bTile[threadIdx.x][threadIdx.y] = b[(threadIdx.x + k) * N + col];
    __syncthreads();

    for(int i = 0; i < BLOCK_SIZE; i++){
      sum += aTile[threadIdx.x][i] * bTile[i][threadIdx.y];
    }
    __syncthreads();
  }
  c[row * N + col] = sum;
}

__global__ void matrixMulGPU( int * a, int * b, int * c )
{
  int val = 0;

  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < N && col < N)
  {
    for ( int k = 0; k < N; ++k )
      val += a[row * N + k] * b[k * N + col];
    c[row * N + col] = val;
  }
}

void matrixMulCPU( int * a, int * b, int * c )
{
  int val = 0;

  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      val = 0;
      for ( int k = 0; k < N; ++k )
        val += a[row * N + k] * b[k * N + col];
      c[row * N + col] = val;
    }
}

int main()
{
  int *a, *b, *c_cpu, *c_gpu;

  int size = N * N * sizeof (int); // Number of bytes of an N x N matrix

  // Allocate memory
  hipMallocManaged (&a, size);
  hipMallocManaged (&b, size);
  hipMallocManaged (&c_cpu, size);
  hipMallocManaged (&c_gpu, size);

  // Initialize memory
  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      a[row*N + col] = row;
      b[row*N + col] = col+2;
      c_cpu[row*N + col] = 0;
      c_gpu[row*N + col] = 0;
    }

  dim3 threads_per_block (8, 8, 1); // A 8 * 8 block threads
  dim3 number_of_blocks ((N + threads_per_block.x - 1)/ threads_per_block.x, (N + threads_per_block.y - 1) / threads_per_block.y, 1);

  matrixMulGPU <<< number_of_blocks, threads_per_block >>> ( a, b, c_gpu );

  hipDeviceSynchronize(); // Wait for the GPU to finish before proceeding

  // Call the CPU version to check our work
  matrixMulCPU( a, b, c_cpu );

  // Compare the two answers to make sure they are equal
  bool error = false;
  for( int row = 0; row < N && !error; ++row )
    for( int col = 0; col < N && !error; ++col )
      if (c_cpu[row * N + col] != c_gpu[row * N + col])
      {
        printf("FOUND ERROR at c[%d][%d]\n", row, col);
        error = true;
        break;
      }
  if (!error)
    printf("Success!\n");

  // Free all our allocated memory
  hipFree(a); hipFree(b);
  hipFree( c_cpu ); hipFree( c_gpu );
}
