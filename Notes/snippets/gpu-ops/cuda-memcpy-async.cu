int N = 2<<24;
int size = N * sizeof(int);

int *host_array;
int *device_array;

hipHostMalloc(&host_array, size);               // Pinned host memory allocation.
hipMalloc(&device_array, size);                 // Allocation directly on the active GPU device.

initializeData(host_array, N);                   // Assume this application needs to initialize on the host.

const int numberOfSegments = 4;                  // This example demonstrates slicing the work into 4 segments.
int segmentN = N / numberOfSegments;             // A value for a segment's worth of `N` is needed.
size_t segmentSize = size / numberOfSegments;    // A value for a segment's worth of `size` is needed.

// For each of the 4 segments...
for (int i = 0; i < numberOfSegments; ++i)
{
  // Calculate the index where this particular segment should operate within the larger arrays.
  segmentOffset = i * segmentN;

  // Create a stream for this segment's worth of copy and work.
  hipStream_t stream;
  hipStreamCreate(&stream);

  // Asynchronously copy segment's worth of pinned host memory to device over non-default stream.
  hipMemcpyAsync(&device_array[segmentOffset],  // Take care to access correct location in array.
                  &host_array[segmentOffset],    // Take care to access correct location in array.
                  segmentSize,                   // Only copy a segment's worth of memory.
                  hipMemcpyHostToDevice,
                  stream);                       // Provide optional argument for non-default stream.

  // Execute segment's worth of work over same non-default stream as memory copy.
  kernel<<<number_of_blocks, threads_per_block, 0, stream>>>(&device_array[segmentOffset], segmentN);

  // `hipStreamDestroy` will return immediately (is non-blocking), but will not actually destroy stream until
  // all stream operations are complete.
  hipStreamDestroy(stream);
}