// processArrayWithDivergence took 0.180000 milliseconds
// processArrayWithoutDivergence took 0.018368 milliseconds
// ncu --set full divergence

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void processArrayWithDivergence(int *data, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        if (data[idx] % 2 == 0) {
            data[idx] = data[idx] * 2;
        } else {
            data[idx] = data[idx] + 1;
        }
    }
}

__global__ void processArrayWithoutDivergence(int *data, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        int isEven = !(data[idx] % 2);
        data[idx] = isEven * (data[idx] * 2) + (!isEven) * (data[idx] + 1);
    }
}

void benchmarkKernel(void (*kernel)(int *, int), int *data, int N, const char *kernelName) {
    int *devData;
    hipMalloc(&devData, N * sizeof(int));
    hipMemcpy(devData, data, N * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEventRecord(start);
    kernel<<<blocksPerGrid, threadsPerBlock>>>(devData, N);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%s took %f milliseconds\n", kernelName, milliseconds);

    hipMemcpy(data, devData, N * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(devData);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    const int N = 1 << 20; // Example size
    int *data = (int *)malloc(N * sizeof(int));

    // Initialize data
    for(int i = 0; i < N; i++) {
        data[i] = i;
    }

    benchmarkKernel(processArrayWithDivergence, data, N, "processArrayWithDivergence");
    benchmarkKernel(processArrayWithoutDivergence, data, N, "processArrayWithoutDivergence");

    free(data);
    return 0;
}
