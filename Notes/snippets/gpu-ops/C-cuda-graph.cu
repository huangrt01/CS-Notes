
*** record

hipGraph_t graph;
hipStreamBeginCapture(stream);
... CUDA calls on stream
hipStreamEndCapture(stream, &graph);

*** make CPU code async

hipGraph_t graph;
hipStreamBeginCapture(a);
kernel1<<<,,,a>>>();
hipEventRecord(e1, a);
kernel2<<<,,,b>>>();
hipStreamWaitEvent(b, e1);
hipMemcpyAsync(,,,,b);
kernel3<<<,,,a>>>();
hipEventRecord(e3, a);

hipLaunchHostFunc(b, cpucode, params);

hipStreamWaitEvent(b, e3);
kernel4<<<,,,b>>>();
hipStreamEndCapture(a, &graph);



*** explicit

hipGraph_t graph;
hipGraphCreate(&graph, 0);
hipGraphNode_t k1,k2,k3,k4,mc,cpu;


hipGraphAddKernelNode(&k1, graph,
0, 0, // no dependency yet
paramsK1, 0);
...
hipGraphAddKernelNode(&k4, graph,
0, 0, paramsK4, 0);
hipGraphAddMemcpyNode(&mc, graph,
0, 0, paramsMC);
hipGraphAddHostNode(&cpu, graph,
0, 0, paramsCPU);


__host__ hipError_t
hipGraphAddKernelNode(hipGraphNode_t* pGraphNode,
	hipGraph_t graph, const hipGraphNode_t* pDependencies,
	size_t numDependencies, const hipKernelNodeParams* pNodeParams);


struct hipKernelNodeParams
{
	void* func; // Kernel function
	dim3 gridDim;
	dim3 blockDim;
	unsigned int sharedMemBytes;
	void **kernelParams; // Array of pointers to arguments
	void **extra; // (low-level alternative to kernelParams)
};

hipGraphAddDependencies(graph,
	&k1, &k3, 1); // kernel1 -> kernel3
hipGraphAddDependencies(graph,
	&k1, &mc, 1); // kernel1 -> memcpy
hipGraphAddDependencies(graph,
	&k2, &mc, 1); // kernel2 -> memcpy
hipGraphAddDependencies(graph,
	&mc, &cpu, 1); // memcpy -> cpu
hipGraphAddDependencies(graph,
	&k3, &k4, 1); // kernel3 -> kernel4
hipGraphAddDependencies(graph,
	&cpu, &k4, 1); // cpu -> kernel4

// Instantiating and running the graph
hipGraphExec_t exec;
hipGraphInstantiate(&exec, graph, 0, 0, 0);
hipGraphLaunch(exec, stream);
hipStreamSynchronize(stream);
	
- Once a graph is instantiated, its topology cannot be changed
- Kernel/memcpy/call… parameters can still be changed using hipGraphExecUpdate
	or cudaGraphExec{Kernel,Host,Memcpy,Memset}NodeSetParams