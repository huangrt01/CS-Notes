#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#include <assert.h>

#define SOFTENING 1e-9f

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct { float x, y, z, vx, vy, vz; } Body;

void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */

__global__ 
void bodyForce(Body *p, float dt, int n) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }

    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}

__global__
void updatePos(Body *p, float dt, int n){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) { // integrate position
    p[i].x += p[i].vx*dt;
    p[i].y += p[i].vy*dt;
    p[i].z += p[i].vz*dt;
  }
}

int main(const int argc, const char** argv) {

  /*
   * Do not change the value for `nBodies` here. If you would like to modify it,
   * pass values into the command line.
   */

  int nBodies = 2<<11;
  int salt = 0;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);

  /*
   * This salt is for assessment reasons. Tampering with it will result in automatic failure.
   */

  if (argc > 2) salt = atoi(argv[2]);

  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations

  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  printf("Device ID: %d\tNumber of SMs: %d\n", deviceId, numberOfSMs);

  int bytes = nBodies * sizeof(Body);
  float *buf;

  hipMallocManaged(&buf, bytes);

  Body *p = (Body*)buf;

  /*
   * As a constraint of this exercise, `randomizeBodies` must remain a host function.
   */

  hipMemPrefetchAsync(buf, bytes, hipCpuDeviceId);

  randomizeBodies(buf, 6 * nBodies); // Init pos / vel data

  double totalTime = 0.0;

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;


  /*******************************************************************/
  // Do not modify these 2 lines of code.
  for (int iter = 0; iter < nIters; iter++) {
    StartTimer();
  /*******************************************************************/

    hipMemPrefetchAsync(buf, bytes, deviceId);

    bodyForce<<<numberOfBlocks, threadsPerBlock>>>(p, dt, nBodies); // compute interbody forces

    checkCuda(hipGetLastError());

    checkCuda(hipDeviceSynchronize());

    updatePos<<<numberOfBlocks, threadsPerBlock>>>(p, dt, nBodies);

    checkCuda(hipGetLastError());

    checkCuda(hipDeviceSynchronize());

  /*******************************************************************/
  // Do not modify the code in this section.
    const double tElapsed = GetTimer() / 1000.0;
    totalTime += tElapsed;
  }

  double avgTime = totalTime / (double)(nIters);
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
  checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
  checkAccuracy(buf, nBodies);
  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
  salt += 1;
#endif
  /*******************************************************************/

  hipFree(buf);
}
