
*** Specify dependencies between tasks

hipEvent_t e;
hipEventCreate(&e);
kernel1<<<,,,a>>>();
hipEventRecord(e, a);
hipStreamWaitEvent(b, e);
kernel2<<<,,,b>>>();
hipEventDestroy(e);


*** measure timing

hipEventRecord(start, 0);
kernel<<<>>>();
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
float elapsedTime;
hipEventElapsedTime(&elapsedTime, start, stop);