#include "hip/hip_runtime.h"
// MIT License

// Copyright (c) 2024 Andrej Karpathy

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.


#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <nvbench/nvbench.cuh>
#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/host_vector.h>
#include <thrust/random.h>
#include <cub/block/block_reduce.cuh>
// kernel2: 没做kernel fusion, 传统reductions实现(对标reduce6，但没用warpReduce，也没有unrolling)
// |         64 |   7744x |  81.841 us | 1.69% |  64.575 us | 1.32% | 780.538 GB/s | 19.40% |   9366x |  53.487 us |
// kernel3: 一个warp处理一次归一化
// |        128 |  14736x | 48.620 us | 2.34% | 33.964 us | 1.77% |   1.484 TB/s | 36.89% |  20972x | 23.842 us |
// kernel4: block_size = 64, 一个block处理一次归一化，利用hipcub::BlockReduce拆的更细，并且比cg::reduce简洁
// |         64 |  16304x |  45.328 us | 2.17% |  30.688 us | 1.20% |   1.642 TB/s | 40.83% |  25130x |  19.897 us |

__global__ void mean_kernel(float* mean, const float* inp, int N, int C, int block_size) {
    extern __shared__ float shared[];
    int idx = blockIdx.x; // range [0, B*T)
    int tid = threadIdx.x; // range [0, block_size)
    const float* x = inp + idx * C;
    // thread coarsening
    float sum = 0.0f;
    for (int i = tid; i < C; i += block_size) {
        sum += x[i];
    }
    shared[tid] = sum;
    __syncthreads();
    // reductions
    for (int stride = block_size / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
    }
    // write the final result (at thread 0) to global memory
    if (tid == 0) {
        mean[idx] = shared[0] / C;
    }
}

__global__ void rstd_kernel(float* rstd, const float* inp, const float* mean, int N, int C, int block_size) {
    extern __shared__ float shared[];
    int idx = blockIdx.x; // range [0, B*T)
    int tid = threadIdx.x; // range [0, block_size)
    const float* x = inp + idx * C;
    float m = mean[idx];
    // thread coarsening
    float sum = 0.0f;
    for (int i = tid; i < C; i += block_size) {
        float diff = x[i] - m;
        sum += diff * diff;
    }
    shared[tid] = sum;
    __syncthreads();
    // reductions
    for (int stride = block_size / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
    }
    // write the final result (at thread 0) to global memory
    if (tid == 0) {
        rstd[idx] = 1.0f / sqrtf(shared[0] / C + 1e-5f);
    }
}

__global__ void normalization_kernel(float* out, const float* inp, float* mean, float* rstd,
                                     const float* weight, const float* bias, int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int bt = idx / C;
    int c = idx % C;

    float m = mean[bt];
    float s = rstd[bt];
    float xi = inp[idx];
    float n = s * (xi - m);
    float o = n * weight[c] + bias[c];

    out[idx] = o;
}

void kernel2(nvbench::state &state)
{
  int B = 8;
  int T = 1024;
  int C = 768;

  thrust::host_vector<float> h_inp(B * T * C);
  thrust::host_vector<float> h_weight(C);
  thrust::host_vector<float> h_bias(C);

  thrust::default_random_engine gen(42);
  thrust::uniform_real_distribution<float> dis(-1.0f, 1.0f);
  thrust::generate(h_inp.begin(), h_inp.end(), [&] { return dis(gen); });
  thrust::generate(h_weight.begin(), h_weight.end(), [&] { return dis(gen); });
  thrust::generate(h_bias.begin(), h_bias.end(), [&] { return dis(gen); });

  thrust::device_vector<float> d_out(B * T * C);
  thrust::device_vector<float> d_mean(B * T);
  thrust::device_vector<float> d_rstd(B * T);
  thrust::device_vector<float> d_inp(h_inp);
  thrust::device_vector<float> d_weight(h_weight);
  thrust::device_vector<float> d_bias(h_bias);

  const int N = B * T;
  const int block_size = state.get_int64("block_size");

  state.add_global_memory_reads<float>(d_inp.size() + d_weight.size() + d_bias.size());
  state.add_global_memory_writes<float>(d_out.size() + d_mean.size() + d_rstd.size());

  const int normalization_block_size = 256;
  const int normalization_grid_size =
      (B * T * C + normalization_block_size - 1) / normalization_block_size;

  state.exec([&](nvbench::launch &launch) {
    hipStream_t stream = launch.get_stream();
    mean_kernel<<<B * T, block_size, block_size * sizeof(float)>>>(
      thrust::raw_pointer_cast(d_mean.data()), 
      thrust::raw_pointer_cast(d_inp.data()), 
      N, C, block_size);
    rstd_kernel<<<B * T, block_size, block_size * sizeof(float)>>>(
      thrust::raw_pointer_cast(d_rstd.data()), 
      thrust::raw_pointer_cast(d_inp.data()), 
      thrust::raw_pointer_cast(d_mean.data()), 
      N, C, block_size);
    normalization_kernel<<<normalization_grid_size, normalization_block_size>>>(
      thrust::raw_pointer_cast(d_out.data()), 
      thrust::raw_pointer_cast(d_inp.data()), 
      thrust::raw_pointer_cast(d_mean.data()), 
      thrust::raw_pointer_cast(d_rstd.data()), 
      thrust::raw_pointer_cast(d_weight.data()), 
      thrust::raw_pointer_cast(d_bias.data()), 
      B, T, C);
  });
}

__global__ void layernorm_forward_kernel3(float* __restrict__ out, float* __restrict__ mean, float* __restrict__ rstd,
                                    const float*  __restrict__ inp, const float*  __restrict__ weight,
                                    const float* __restrict__ bias, int N, int C) {
    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    if(idx >= N) {
        return;
    }

    // the row of input that this group of threads is responsible for
    const float* x = inp + idx * C;

    // mean
    float sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        sum += x[i];
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float m = sum / C;
    if(warp.thread_rank() == 0 && mean != nullptr) {
        __stcs(mean + idx, m);
    }

    // rstd
    sum = 0.0f;
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float diff = x[i] - m;
        sum += diff * diff;
    }
    sum = cg::reduce(warp, sum, cg::plus<float>{});
    float s = rsqrtf(sum / C + 1e-5f);
    if(warp.thread_rank() == 0 && rstd != nullptr) {
        __stcs(rstd + idx, s);
    }

    // final normalization and scaling by weight/bias
    float* o = out + idx * C;
    for (int c = warp.thread_rank(); c < C; c += warp.size()) {
        // load and store using the .cs "streaming" hint to the compiler,
        // indicating that this data will not be reused soon, and can be streamed through the caches
        // this allows the threads to get more cache-hits for the (shared) weight and bias parameters
        float n = s * (__ldcs(x+c) - m);
        __stcs(o+c, n * weight[c] + bias[c]);
    }
}


void kernel3(nvbench::state &state)
{
  int B = 8;
  int T = 1024;
  int C = 768;

  thrust::host_vector<float> h_inp(B * T * C);
  thrust::host_vector<float> h_weight(C);
  thrust::host_vector<float> h_bias(C);

  thrust::default_random_engine gen(42);
  thrust::uniform_real_distribution<float> dis(-1.0f, 1.0f);
  thrust::generate(h_inp.begin(), h_inp.end(), [&] { return dis(gen); });
  thrust::generate(h_weight.begin(), h_weight.end(), [&] { return dis(gen); });
  thrust::generate(h_bias.begin(), h_bias.end(), [&] { return dis(gen); });

  thrust::device_vector<float> d_out(B * T * C);
  thrust::device_vector<float> d_mean(B * T);
  thrust::device_vector<float> d_rstd(B * T);
  thrust::device_vector<float> d_inp(h_inp);
  thrust::device_vector<float> d_weight(h_weight);
  thrust::device_vector<float> d_bias(h_bias);

  const int N = B * T;
  const int block_size = state.get_int64("block_size");
  const int grid_size = (N * 32 + block_size - 1) / block_size;

  state.add_global_memory_reads<float>(d_inp.size() + d_weight.size() + d_bias.size());
  state.add_global_memory_writes<float>(d_out.size() + d_mean.size() + d_rstd.size());

  state.exec([&](nvbench::launch &launch) {
    hipStream_t stream = launch.get_stream();
    layernorm_forward_kernel3<<<grid_size, block_size, 0, stream>>>(
      thrust::raw_pointer_cast(d_out.data()), 
      thrust::raw_pointer_cast(d_mean.data()), 
      thrust::raw_pointer_cast(d_rstd.data()), 
      thrust::raw_pointer_cast(d_inp.data()), 
      thrust::raw_pointer_cast(d_weight.data()),
      thrust::raw_pointer_cast(d_bias.data()),
      N, C);
  });
}

template<int BlockSize>
__global__ __launch_bounds__(BlockSize)
void layernorm_forward_kernel4(float* __restrict__ out, float* __restrict__ mean, float* __restrict__ rstd,
    const float*  __restrict__ inp, const float*  __restrict__ weight,
    const float* __restrict__ bias, int N, int C) {
    int tid = threadIdx.x;
    int idx = blockIdx.x;
    if(idx >= N) {
        return;
    }

    // the row of input that this group of threads is responsible for
    const float* x = inp + idx * C;

    // mean
    float sum = 0.0f;
    for (int i = tid; i < C; i += BlockSize) {
        sum += x[i];
    }
    sum = hipcub::BlockReduce<float, BlockSize>().Sum(sum);
    __shared__ float shared_mean;
    if(tid == 0 && mean != nullptr) {
        float m = sum / C;
        shared_mean = m;
        __stcs(mean + idx, m);
    }
    __syncthreads();
    const float m = shared_mean;

    // rstd
    sum = 0.0f;
    for (int i = tid; i < C; i += BlockSize) {
        float diff = x[i] - m;
        sum += diff * diff;
    }
    sum = hipcub::BlockReduce<float, BlockSize>().Sum(sum);
    __shared__ float shared_s;
    if(tid == 0 && rstd != nullptr) {
        float s_val = rsqrtf(sum / C + 1e-5f);
        shared_s = s_val;
        __stcs(rstd + idx, s_val);
    }
    __syncthreads();
    const float s = shared_s;

    // final normalization and scaling by weight/bias
    float* o = out + idx * C;
    for (int i = tid; i < C; i += BlockSize) {
        // load and store using the.cs "streaming" hint to the compiler,
        float n = s * (__ldcs(x+i) - m);
        __stcs(o+i, n * weight[i] + bias[i]);
    }
}

void kernel4(nvbench::state &state)
{
  int B = 8;
  int T = 1024;
  int C = 768;

  thrust::host_vector<float> h_inp(B * T * C);
  thrust::host_vector<float> h_weight(C);
  thrust::host_vector<float> h_bias(C);

  thrust::default_random_engine gen(42);
  thrust::uniform_real_distribution<float> dis(-1.0f, 1.0f);
  thrust::generate(h_inp.begin(), h_inp.end(), [&] { return dis(gen); });
  thrust::generate(h_weight.begin(), h_weight.end(), [&] { return dis(gen); });
  thrust::generate(h_bias.begin(), h_bias.end(), [&] { return dis(gen); });

  thrust::device_vector<float> d_out(B * T * C);
  thrust::device_vector<float> d_mean(B * T);
  thrust::device_vector<float> d_rstd(B * T);
  thrust::device_vector<float> d_inp(h_inp);
  thrust::device_vector<float> d_weight(h_weight);
  thrust::device_vector<float> d_bias(h_bias);

  const int N = B * T;
  const int current_block_size = state.get_int64("block_size");
  const int grid_size = N; // Kernel4 processes N items, one per block

  state.add_global_memory_reads<float>(d_inp.size() + d_weight.size() + d_bias.size());
  state.add_global_memory_writes<float>(d_out.size() + d_mean.size() + d_rstd.size());

  state.exec([&](nvbench::launch &launch) {
    hipStream_t stream = launch.get_stream();
    switch (current_block_size) {
        case 32:
            layernorm_forward_kernel4<32><<<grid_size, 32, 0, stream>>>(
                thrust::raw_pointer_cast(d_out.data()), thrust::raw_pointer_cast(d_mean.data()), thrust::raw_pointer_cast(d_rstd.data()),
                thrust::raw_pointer_cast(d_inp.data()), thrust::raw_pointer_cast(d_weight.data()), thrust::raw_pointer_cast(d_bias.data()),
                N, C);
            break;
        case 64:
            layernorm_forward_kernel4<64><<<grid_size, 64, 0, stream>>>(
                thrust::raw_pointer_cast(d_out.data()), thrust::raw_pointer_cast(d_mean.data()), thrust::raw_pointer_cast(d_rstd.data()),
                thrust::raw_pointer_cast(d_inp.data()), thrust::raw_pointer_cast(d_weight.data()), thrust::raw_pointer_cast(d_bias.data()),
                N, C);
            break;
        case 128:
            layernorm_forward_kernel4<128><<<grid_size, 128, 0, stream>>>(
                thrust::raw_pointer_cast(d_out.data()), thrust::raw_pointer_cast(d_mean.data()), thrust::raw_pointer_cast(d_rstd.data()),
                thrust::raw_pointer_cast(d_inp.data()), thrust::raw_pointer_cast(d_weight.data()), thrust::raw_pointer_cast(d_bias.data()),
                N, C);
            break;
        case 256:
            layernorm_forward_kernel4<256><<<grid_size, 256, 0, stream>>>(
                thrust::raw_pointer_cast(d_out.data()), thrust::raw_pointer_cast(d_mean.data()), thrust::raw_pointer_cast(d_rstd.data()),
                thrust::raw_pointer_cast(d_inp.data()), thrust::raw_pointer_cast(d_weight.data()), thrust::raw_pointer_cast(d_bias.data()),
                N, C);
            break;
        case 512:
            layernorm_forward_kernel4<512><<<grid_size, 512, 0, stream>>>(
                thrust::raw_pointer_cast(d_out.data()), thrust::raw_pointer_cast(d_mean.data()), thrust::raw_pointer_cast(d_rstd.data()),
                thrust::raw_pointer_cast(d_inp.data()), thrust::raw_pointer_cast(d_weight.data()), thrust::raw_pointer_cast(d_bias.data()),
                N, C);
            break;
        case 1024:
            layernorm_forward_kernel4<1024><<<grid_size, 1024, 0, stream>>>(
                thrust::raw_pointer_cast(d_out.data()), thrust::raw_pointer_cast(d_mean.data()), thrust::raw_pointer_cast(d_rstd.data()),
                thrust::raw_pointer_cast(d_inp.data()), thrust::raw_pointer_cast(d_weight.data()), thrust::raw_pointer_cast(d_bias.data()),
                N, C);
            break;
        default:
            break;
    }
  });
}

NVBENCH_BENCH(kernel2).add_int64_axis("block_size", {32, 64, 128, 256, 512, 1024});
NVBENCH_BENCH(kernel3).add_int64_axis("block_size", {32, 64, 128, 256, 512, 1024});
NVBENCH_BENCH(kernel4).add_int64_axis("block_size", {32, 64, 128, 256, 512, 1024});
