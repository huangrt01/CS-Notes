#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#define N 2048 * 2048 // Number of elements in each vector

/*
 * Aim to profile `saxpy` (without modifying `N`) running under
 * 20us.
 */

__global__ void saxpy(int * a, int * b, int * c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for ( int i = tid; i < N; i += stride ){
        c[i] = 2 * a[i] + b[i];
    }
}

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

int main()
{
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    printf("Device ID: %d\tNumber of SMs: %d\n", deviceId, numberOfSMs);
    
    int *a, *b, *c;

    int size = N * sizeof (int); // The total number of bytes per vector

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&c, size);
    
    hipMemPrefetchAsync(a, size, hipCpuDeviceId);
    hipMemPrefetchAsync(b, size, hipCpuDeviceId);
    hipMemPrefetchAsync(c, size, hipCpuDeviceId);

    // Initialize memory
    for( int i = 0; i < N; ++i )
    {
        a[i] = 2;
        b[i] = 1;
        c[i] = 0;
    }
    
    
    int threads_per_block = 256;
    int number_of_blocks = 32 * numberOfSMs;
    
    hipMemPrefetchAsync(a, size, deviceId);
    hipMemPrefetchAsync(b, size, deviceId);
    hipMemPrefetchAsync(c, size, deviceId);

    saxpy <<< number_of_blocks, threads_per_block >>> ( a, b, c );
    hipDeviceSynchronize();
    
    hipMemPrefetchAsync(c, size, hipCpuDeviceId);
    // Print out the first and last 5 values of c for a quality check
    for( int i = 0; i < 5; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");
    for( int i = N-5; i < N; ++i )
        printf("c[%d] = %d, ", i, c[i]);
    printf ("\n");

    hipFree( a ); hipFree( b ); hipFree( c );
}

