#include "hip/hip_runtime.h"
*** stream priority

https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__STREAM.html#group__CUDART__STREAM_1ge2be9e9858849bf62ba4a8b66d1c3540

hipStreamCreateWithPriority
hipDeviceGetStreamPriorityRange


*** hipStreamQuery

hipStreamQuery 是 CUDA Runtime API 中的一个重要函数，它的核心功能是 以非阻塞的方式查询一个 CUDA 流（Stream）的状态 。

*** example

#include <stdio.h>

__global__
void initWith(float num, float *a, int N)
{

  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    a[i] = num;
  }
}

__global__
void addVectorsInto(float *result, float *a, float *b, int N)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for(int i = index; i < N; i += stride)
  {
    result[i] = a[i] + b[i];
  }
}

void checkElementsAre(float target, float *vector, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(vector[i] != target)
    {
      printf("FAIL: vector[%d] - %0.0f does not equal %0.0f\n", i, vector[i], target);
      exit(1);
    }
  }
  printf("Success! All values calculated correctly.\n");
}

int main()
{
  int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

  const int N = 2<<24;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);

  hipMemPrefetchAsync(a, size, deviceId);
  hipMemPrefetchAsync(b, size, deviceId);
  hipMemPrefetchAsync(c, size, deviceId);

  size_t threadsPerBlock;
  size_t numberOfBlocks;

  threadsPerBlock = 256;
  numberOfBlocks = 32 * numberOfSMs;

  hipError_t addVectorsErr;
  hipError_t asyncErr;

  /*
   * Create 3 streams to run initialize the 3 data vectors in parallel.
   */

  hipStream_t stream1, stream2, stream3;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);
  hipStreamCreate(&stream3);

  /*
   * Give each `initWith` launch its own non-standard stream.
   */

  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream1>>>(3, a, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream2>>>(4, b, N);
  initWith<<<numberOfBlocks, threadsPerBlock, 0, stream3>>>(0, c, N);

  addVectorsInto<<<numberOfBlocks, threadsPerBlock>>>(c, a, b, N);

  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  hipMemPrefetchAsync(c, size, hipCpuDeviceId);

  checkElementsAre(7, c, N);

  /*
   * Destroy streams when they are no longer needed.
   */

  hipStreamDestroy(stream1);
  hipStreamDestroy(stream2);
  hipStreamDestroy(stream3);

  hipFree(a);
  hipFree(b);
  hipFree(c);
}
